#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h>
#include <assert.h>

using bf = __hip_bfloat16;
__device__ inline float to_float(const bf & u) { return __bfloat162float(u); }
__device__ inline bf to_bf(const float & u) { return __float2bfloat16_rn(u); }

typedef bf * __restrict__ F_;

__global__ void forward_kernel(int T, int H, F_ w_, F_ q_, F_ k_, F_ v_, F_ a_, F_ b_, bf* y_, float* s_, float* sa_) {
    constexpr int C = _C_;
    int bind = blockIdx.y, hind = blockIdx.x, i = threadIdx.x;

    float state[C] = {0};
    __shared__ float q[C], k[C], w[C], a[C], b[C];

    for (int t = 0; t < T; t++) {
        int ind = bind*T*H*C + t*H*C + hind * C + i;
        __syncthreads();
        q[i] = to_float(q_[ind]);
        w[i] = __expf(-__expf(to_float(w_[ind])));
        k[i] = to_float(k_[ind]);
        a[i] = to_float(a_[ind]);
        b[i] = to_float(b_[ind]);
        __syncthreads();

        float sa = 0;
#pragma unroll
        for (int j = 0; j < C; j++) {
            sa += a[j] * state[j];
        }
        sa_[ind] = sa;

        float v = to_float(v_[ind]);
        float y = 0;
#pragma unroll
        for (int j = 0; j < C; j++) {
            float& s = state[j];
            s = s * w[j] + sa * b[j] + k[j] * v;
            y += s * q[j];
        }
        y_[ind] = to_bf(y);

        if ((t+1)%_CHUNK_LEN_ == 0) {
            int base = (bind*H+hind)*(T/_CHUNK_LEN_)*C*C + (t/_CHUNK_LEN_)*C*C + i;
#pragma unroll
            for (int j = 0; j < C; j++) {
                s_[base + j*C] = state[j];
            }
        }
    }
}

__global__ void backward_kernel(int T, int H, F_ w_, F_ q_, F_ k_, F_ v_, F_ a_, F_ b_, F_ dy_, float * __restrict__ s_, float * __restrict__ sa_, bf* dw_, bf* dq_, bf* dk_, bf* dv_, bf* da_, bf* db_) {
    constexpr int C = _C_;
    int bind = blockIdx.y, hind = blockIdx.x, i = threadIdx.x;

    float stateT[C] = {0};
    __shared__ float w[C], q[C], k[C], v[C], a[C], b[C], dy[C], sa[C], dSb_shared[C];

    extern __shared__ char smem_[];
    float*dstate = (float*)smem_; //[C*(C+1)];

    for (int j = 0; j < C; j++) {
        dstate[i*(C+1)+j] = 0;
    }

    for (int t = T-1; t >= 0; t--) {
        int ind = bind*T*H*C + t*H*C + hind * C + i;
        float bi, ki, dyi, wi;
        __syncthreads();
        q[i] = to_float(q_[ind]);
        float wi_fac = -__expf(to_float(w_[ind]));
        w[i] = wi = __expf(wi_fac);
        k[i] = ki = to_float(k_[ind]);
        a[i] = to_float(a_[ind]);
        b[i] = bi = to_float(b_[ind]);
        v[i] = to_float(v_[ind]);
        dy[i] = dyi = to_float(dy_[ind]);
        sa[i] = sa_[ind];
        __syncthreads();

        if ((t+1)%_CHUNK_LEN_ == 0) {
            int base = (bind*H+hind)*(T/_CHUNK_LEN_)*C*C + (t/_CHUNK_LEN_)*C*C + i*C;
#pragma unroll
            for (int j = 0; j < C; j++) {
                stateT[j] = s_[base + j];
            }
        }

        float dq = 0;
#pragma unroll
        for (int j = 0; j < C; j++) {
            dq += stateT[j]*dy[j];
        }
        dq_[ind] = to_bf(dq);

        float iwi = 1.0f/wi;
        for (int j = 0; j < C; j++) {
            stateT[j] = (stateT[j] - ki*v[j] - bi*sa[j]) * iwi;
            dstate[i*(C+1)+j] += dyi * q[j];
        }

        float dw = 0, dk = 0, dv = 0, db = 0, dSb = 0;
#pragma unroll
        for (int j = 0; j < C; j++) {
            float ds = dstate[j*(C+1)+i];
            dw += ds*stateT[j];
            dk += ds*v[j];
            db += ds*sa[j];
        }
#pragma unroll
        for (int j = 0; j < C; j++) {
            float ds = dstate[i*(C+1)+j];
            dv += ds*k[j];
            dSb += ds*b[j];
        }
        dw_[ind] = to_bf(dw * wi * wi_fac);
        dk_[ind] = to_bf(dk);
        dv_[ind] = to_bf(dv);
        db_[ind] = to_bf(db);

        __syncthreads();
        dSb_shared[i] = dSb;
        __syncthreads();

        float da = 0;
#pragma unroll
        for (int j = 0; j < C; j++) {
            da += stateT[j]*dSb_shared[j];
        }
        da_[ind] = to_bf(da);

        for (int j = 0; j < C; j++) {
            dstate[i*(C+1)+j] = dstate[i*(C+1)+j]*w[j] + dSb * a[j];
        }
    }
}

void cuda_forward(int B, int T, int H, bf*w, bf*q, bf*k, bf*v, bf*z, bf*a, bf*y, float*s, float*sa) {
    forward_kernel<<<dim3(H,B), dim3(_C_)>>>(T,H,w,q,k,v,z,a,y,s,sa);
}
void cuda_backward(int B, int T, int H, bf*w, bf*q, bf*k, bf*v, bf*z, bf*a, bf*dy, float*s, float*sa, bf*dw, bf*dq, bf*dk, bf*dv, bf*dz, bf*da) {
    assert(T%_CHUNK_LEN_ == 0);
    int shared_mem = _C_*(_C_+1)*4;
    assert(!hipFuncSetAttribute(reinterpret_cast<const void*>(backward_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem));
    backward_kernel<<<dim3(H,B), dim3(_C_), shared_mem>>>(T,H,w,q,k,v,z,a,dy,s,sa,dw,dq,dk,dv,dz,da);
}
