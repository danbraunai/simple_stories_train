#include "hip/hip_runtime.h"
#include "tile.cuh"
#include <assert.h>
typedef bf * __restrict__ F_;

constexpr int WARPS = _C_/16;
constexpr int fw_stages = 1, bw_stages = 1;

__global__ void forward_kernel(int T, int H, F_ w_, F_ q_, F_ k_, F_ v_, F_ a_, F_ b_, F_ s0_, bf* y_, bf* s_, bf* sT_) {
    constexpr int C = _C_, K = 16;
    int bi = blockIdx.y, hi = blockIdx.x;
    extern __shared__ char smem_[];
    char*smem = smem_;

    STile *sw_ = (STile*)smem; smem += sizeof(STile)*fw_stages*WARPS;
    STile *sq_ = (STile*)smem; smem += sizeof(STile)*fw_stages*WARPS;
    STile *sk_ = (STile*)smem; smem += sizeof(STile)*fw_stages*WARPS;
    STile *sv_ = (STile*)smem; smem += sizeof(STile)*fw_stages*WARPS;
    STile *sa_ = (STile*)smem; smem += sizeof(STile)*fw_stages*WARPS;
    STile *sb_ = (STile*)smem; smem += sizeof(STile)*fw_stages*WARPS;
    char*share = (char*)smem;

    int stride = H*C;
    int warpi = threadIdx.x/32;

    auto push = [&](int t) {
        int off = bi*T*H*C + t*K*H*C + hi*C + warpi*16;
        int si = t%fw_stages;
        sw_[si*WARPS+warpi] = GTile(w_+off, stride);
        sq_[si*WARPS+warpi] = GTile(q_+off, stride);
        sk_[si*WARPS+warpi] = GTile(k_+off, stride);
        sv_[si*WARPS+warpi] = GTile(v_+off, stride);
        sa_[si*WARPS+warpi] = GTile(a_+off, stride);
        sb_[si*WARPS+warpi] = GTile(b_+off, stride);
    };
    for (int t = 0; t < fw_stages-1 && t < T/K; t++) push(t), __commit_group();

    FTile state[WARPS];
    for (int i = 0; i < WARPS; i++) {
        int off = bi*H*C*C + hi*C*C + warpi*16*C + i*16;
        RTile tmp;
        tmp = GTile(s0_+off, C);
        state[i] = tmp;
    }

    for (int t = 0; t < T/K; t++) {
        __syncthreads();
        if (t+fw_stages-1 < T/K)
            push(t+fw_stages-1);
        __commit_group();
        __wait_groups<fw_stages-1>();
        __syncthreads();
        int si = t%fw_stages;
        STile &sw = sw_[si*WARPS+warpi], &sq = sq_[si*WARPS+warpi], &sk = sk_[si*WARPS+warpi], &sv = sv_[si*WARPS+warpi], &sa = sa_[si*WARPS+warpi], &sb = sb_[si*WARPS+warpi];

        FTile w = (RTile)sw;
        apply_(w, [](float x) { return __expf(-__expf(x)); });
        FTile fw = w;
        FTile non_incl_pref = cumprodv<0,0>(fw);
        FTile incl_pref = non_incl_pref * w;
        FTile inv_incl_pref = incl_pref;
        apply_(inv_incl_pref, [](float x) { return 1.f/x; });

        RTile wq = (RTile)sq *     incl_pref, kwi = (RTile)sk * inv_incl_pref;
        RTile wa = (RTile)sa * non_incl_pref, bwi = (RTile)sb * inv_incl_pref;
        FTile ab = sum_warp<1,WARPS>((float2*)share, tril<1>(wa % bwi));
        RTile ak = sum_warp<1,WARPS>((float2*)share, tril<1>(wa % kwi));

        RTile ab_inv;
        __syncthreads();
        if (threadIdx.x < 32) ab_inv = tri_minv(ab, (float*)share);
        __syncthreads();
        ab_inv = from_warp(ab_inv, 0, (float4*)share);

        RTile vt = sv.t();
        FTile ab_ut = vt % ak;
        for (int i = 0; i < WARPS; i++)
            ab_ut += state[i] % from_warp(wa, i, (float4*)share);
        RTile ut = FTile(ab_ut % ab_inv);

        FTile y = sum_warp<1,WARPS>((float2*)share, tril<0>(wq % kwi)) % vt;
        y +=      sum_warp<1,WARPS>((float2*)share, tril<0>(wq % bwi)) % ut;
        for (int i = 0; i < WARPS; i++)
            y += from_warp(wq, i, (float4*)share) % state[i];

        int off = bi*T*H*C + t*K*H*C + hi*C + warpi*16;
        GTile(y_+off, stride) = RTile(y);

        RTile kwt = transpose(kwi*fw), bwt = transpose(bwi*fw);
        for (int i = 0; i < WARPS; i++) {
            int off = bi*H*(T/K)*C*C + hi*(T/K)*C*C + t*C*C + warpi*16*C + i*16;
            GTile(s_+off, C) = (RTile)state[i];

            FTile fstate = state[i] * from_warp(fw, i, (float4*)share);
            fstate += vt % from_warp(kwt, i, (float4*)share);
            fstate += ut % from_warp(bwt, i, (float4*)share);
            state[i] = fstate;
        }
    }
    for (int i = 0; i < WARPS; i++) {
        int off = bi*H*C*C + hi*C*C + warpi*16*C + i*16;
        GTile(sT_+off, C) = state[i];
    }
}

void cuda_forward(int B, int T, int H, bf*w, bf*q, bf*k, bf*v, bf*z, bf*a, bf*s0, bf*y, bf*s, bf*sT) {
    assert(T%16 == 0);
    constexpr int tmp_size1 = sizeof(float4)*32, tmp_size2 = sizeof(float)*16*16*2;
    constexpr int threads = 32*WARPS, shared_mem = sizeof(STile)*fw_stages*WARPS*6 + (tmp_size1 > tmp_size2 ? tmp_size1 : tmp_size2);
    static int reported = 0;
    if (!reported++) {
#if defined VERBOSE
        printf("forward_kernel() uses %d bytes of (dynamic) shared memory\n", shared_mem);
#endif
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(forward_kernel));
        int cur_mem = attr.maxDynamicSharedSizeBytes;
        if (shared_mem > cur_mem) {
#if defined VERBOSE
            printf("Increasing forward_kernel's MaxDynamicSharedMemorySize from %d to %d\n", cur_mem, shared_mem);
#endif
            assert(!hipFuncSetAttribute(reinterpret_cast<const void*>(forward_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem));
        }
    }
    forward_kernel<<<dim3(H,B), dim3(threads), shared_mem>>>(T,H,w,q,k,v,z,a,s0,y,s,sT);
}


__global__ void backward_kernel(int T, int H, F_ w_, F_ q_, F_ k_, F_ v_, F_ a_, F_ b_, F_ dy_, F_ s_, F_ dsT_, bf* dw_, bf* dq_, bf* dk_, bf* dv_, bf* da_, bf* db_, bf* ds0_) {
    constexpr int C = _C_, K = 16;
    int bi = blockIdx.y, hi = blockIdx.x;
    extern __shared__ char smem_[];
    char*smem = smem_;

    STile *sw_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *sq_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *sk_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *sv_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *sa_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *sb_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *sdy_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *state_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS*WARPS;
    char*share = (char*)smem;

    int stride = H*C;
    int warpi = threadIdx.x/32;

    auto push = [&](int t) {
        int off = bi*T*H*C + t*K*H*C + hi*C + warpi*16;
        int si = t%fw_stages;
        sw_[si*WARPS+warpi] = GTile(w_+off, stride);
        sq_[si*WARPS+warpi] = GTile(q_+off, stride);
        sk_[si*WARPS+warpi] = GTile(k_+off, stride);
        sv_[si*WARPS+warpi] = GTile(v_+off, stride);
        sa_[si*WARPS+warpi] = GTile(a_+off, stride);
        sb_[si*WARPS+warpi] = GTile(b_+off, stride);
        sdy_[si*WARPS+warpi] = GTile(dy_+off, stride);
        for (int i = 0; i < WARPS; i++) {
            int off2 = bi*H*(T/K)*C*C + hi*(T/K)*C*C + t*C*C + warpi*16*C + i*16;
            state_[si*WARPS*WARPS+warpi*WARPS+i] = GTile(s_+off2, C);
        }
    };

    FTile dstate[WARPS];
    for (int i = 0; i < WARPS; i++) {
        int off = bi*H*C*C + hi*C*C + warpi*16*C + i*16;
        RTile tmp;
        tmp = GTile(dsT_+off, C);
        dstate[i] = tmp;
        __commit_group();
    }

    for (int t = 0; t < bw_stages-1 && t < T/K; t++) push(T/K-1-t), __commit_group();

    for (int t = T/K-1; t >= 0; t--) {
        __syncthreads();
        if (t-bw_stages+1 >= 0)
            push(t-bw_stages+1);
        __commit_group();
        __wait_groups<bw_stages-1>();
        __syncthreads();
        int si = t%bw_stages;
        STile &sw = sw_[si*WARPS+warpi], &sq = sq_[si*WARPS+warpi], &sk = sk_[si*WARPS+warpi], &sv = sv_[si*WARPS+warpi], &sa = sa_[si*WARPS+warpi], &sb = sb_[si*WARPS+warpi], &sdy = sdy_[si*WARPS+warpi];
        STile*state = state_+si*WARPS*WARPS;

        FTile w = (RTile)sw;
        apply_(w, [](float x) { return __expf(-__expf(x)); });
        FTile fw = w;
        FTile non_incl_pref = cumprodv<0,0>(fw);
        FTile incl_pref = non_incl_pref * w;
        FTile inv_incl_pref = incl_pref;
        apply_(inv_incl_pref, [](float x) { return 1.f/x; });

        RTile wq = (RTile)sq *     incl_pref, kwi = (RTile)sk * inv_incl_pref;
        RTile wa = (RTile)sa * non_incl_pref, bwi = (RTile)sb * inv_incl_pref;
        FTile ab = sum_warp<1,WARPS>((float2*)share, tril<1>(wa % bwi));
        RTile ak = sum_warp<1,WARPS>((float2*)share, tril<1>(wa % kwi));

        RTile ab_inv;
        __syncthreads();
        if (threadIdx.x < 32) ab_inv = tri_minv(ab, (float*)share);
        __syncthreads();
        ab_inv = from_warp(ab_inv, 0, (float4*)share);

        RTile vt = sv.t();
        FTile ab_ut = vt % ak;
        for (int i = 0; i < WARPS; i++)
            ab_ut += state[warpi*WARPS+i] % from_warp(wa, i, (float4*)share);
        RTile ut = FTile(ab_ut % ab_inv);

        RTile qb = sum_warp<1,WARPS>((float2*)share, tril<0>(wq % bwi));
        RTile qk = sum_warp<1,WARPS>((float2*)share, tril<0>(wq % kwi));

        RTile dyt = sdy.t();
        FTile dut = FTile(dyt % transpose(qb));
        FTile dv = transpose(qk) % dyt;
        for (int i = 0; i < WARPS; i++) {
            RTile dstatei = dstate[i];
            dut += dstatei % from_warp(bwi*fw, i, (float4*)share);
            dv += from_warp(kwi*fw, i, (float4*)share) % dstatei;
        }
        RTile dab_ut = FTile(dut % transpose(ab_inv));
        dv += transpose(ak) % dab_ut;

        int off = bi*T*H*C + t*K*H*C + hi*C + warpi*16;
        GTile(dv_+off, stride) = RTile(dv);

        FTile dab = sum_warp<1,WARPS>((float2*)share, tril<1>(transpose(dab_ut) % transpose(ut)));
        FTile dak = sum_warp<1,WARPS>((float2*)share, tril<1>(transpose(dab_ut) % transpose(vt)));
        FTile dab_u_state0;
        dab_u_state0.zero_();
        for (int i = 0; i < WARPS; i++)
            dab_u_state0 += from_warp(transpose(dab_ut), i, (float4*)share) % state[i*WARPS+warpi].t();

        FTile da = dab_u_state0;
        da += dab % transpose(bwi);
        da += dak % transpose(kwi);
        da = non_incl_pref * da;
        GTile(da_+off, stride) = RTile(da);

        FTile dqb = sum_warp<1,WARPS>((float2*)share, tril<0>(transpose(dyt) % transpose(ut)));
        FTile dqk = sum_warp<1,WARPS>((float2*)share, tril<0>(transpose(dyt) % transpose(vt)));
        FTile dy_state0;
        dy_state0.zero_();
        for (int i = 0; i < WARPS; i++)
            dy_state0 += from_warp(transpose(dyt), i, (float4*)share) % state[i*WARPS+warpi].t();

        FTile dq = dy_state0;
        dq += dqb % transpose(bwi);
        dq += dqk % transpose(kwi);
        dq = incl_pref * dq;
        GTile(dq_+off, stride) = RTile(dq);

        RTile wqt = transpose(wq), wat = transpose(wa);

        FTile u_dstate, v_dstate, dw;
        u_dstate.zero_();
        v_dstate.zero_();
        dw.zero_();
        RTile ones;
        for (int i = 0; i < 4; i++) ones.data[i] = to_bf2({1.f,1.f});
        for (int i = 0; i < WARPS; i++) {
            int tid = threadIdx.x%32;
            if (warpi == i) {
                for (int j = 0; j < WARPS; j++) {
                    RTile ra = dstate[j];
                    ((float4*)share)[j*32+tid] = *((float4*)ra.data);
                }
            }
            RTile dstatei;// = dstate[i*WARPS+warpi];
            __syncthreads();
            *((float4*)dstatei.data) = ((float4*)share)[warpi*32+tid];
            __syncthreads();
            RTile dstatei_t = transpose(dstatei);
            v_dstate += from_warp(transpose(vt), i, (float4*)share) % dstatei_t;
            u_dstate += from_warp(transpose(ut), i, (float4*)share) % dstatei_t;
            dw += ones % transpose((RTile)state[i*WARPS+warpi]*dstatei);
        }

        FTile db = fw * u_dstate;
        db += transpose(dab) % wat;
        db += transpose(dqb) % wqt;
        db = inv_incl_pref * db;
        GTile(db_+off, stride) = RTile(db);

        FTile dk = fw * v_dstate;
        dk += transpose(dak) % wat;
        dk += transpose(dqk) % wqt;
        dk = inv_incl_pref * dk;
        GTile(dk_+off, stride) = RTile(dk);

        dw = fw * dw;
        dw += fast_dw<1>(dab,wa,bwi);
        dw += fast_dw<1>(dak,wa,kwi);
        dw += fast_dw<0>(dqb,wq,bwi);
        dw += fast_dw<0>(dqk,wq,kwi);
        FTile tmp;
        dw += cumsumv<0,0>(tmp = v_dstate*(fw*kwi));
        dw += cumsumv<0,0>(tmp = u_dstate*(fw*bwi));
        dw += cumsumv<0,1>(tmp = dab_u_state0*wa);
        dw += cumsumv<1,1>(tmp = dy_state0*wq);

        FTile dw_fac = (RTile)sw;
        apply_(dw_fac, [](float x) { return -__expf(x); });
        dw = dw * dw_fac;
        GTile(dw_+off, stride) = RTile(dw);

        __syncthreads();
        for (int i = 0; i < WARPS; i++) {
            FTile ndstate = dstate[i] * from_warp(fw, i, (float4*)share);
            ndstate += dyt % from_warp(wqt, i, (float4*)share);
            ndstate += dab_ut % from_warp(wat, i, (float4*)share);
            dstate[i] = ndstate;
        }
        __syncthreads();
    }
    for (int i = 0; i < WARPS; i++) {
        int off = bi*H*C*C + hi*C*C + warpi*16*C + i*16;
        GTile(ds0_+off, C) = dstate[i];
    }
}

void cuda_backward(int B, int T, int H, bf*w, bf*q, bf*k, bf*v, bf*z, bf*a, bf*dy, bf*s, bf*dsT, bf*dw, bf*dq, bf*dk, bf*dv, bf*dz, bf*da, bf*ds0) {
    assert(T%16 == 0);
    constexpr int tmp_size1 = sizeof(float4)*32*WARPS, tmp_size2 = sizeof(float)*16*16*2;
    constexpr int threads = 32*WARPS, shared_mem = sizeof(STile)*WARPS*bw_stages*(7+WARPS) + (tmp_size1 > tmp_size2 ? tmp_size1 : tmp_size2);
    static int reported = 0;
    if (!reported++) {
#if defined VERBOSE
        printf("backward_kernel() uses %d bytes of (dynamic) shared memory\n", shared_mem);
#endif
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(backward_kernel));
        int cur_mem = attr.maxDynamicSharedSizeBytes;
        if (shared_mem > cur_mem) {
#if defined VERBOSE
            printf("Increasing backward_kernel's MaxDynamicSharedMemorySize from %d to %d\n", cur_mem, shared_mem);
#endif
            assert(!hipFuncSetAttribute(reinterpret_cast<const void*>(backward_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem));
        }
    }
    backward_kernel<<<dim3(H,B), dim3(threads), shared_mem>>>(T,H,w,q,k,v,z,a,dy,s,dsT,dw,dq,dk,dv,dz,da,ds0);
}
